#include "hip/hip_runtime.h"
﻿// lab5host.cpp : Defines the entry point for the console application.
//

#include "stdafx.h"
#include <math.h>
#include <stdio.h>
#include <chrono>
#include "tga.h"
#define N 8

void calculate_matrix(double matrix[N * N]);
void transpose_matrix(double matrix[N * N], double tranposed[N * N]);

int main()
{
	unsigned char* data;

	unsigned char* result_data;
	unsigned width, height, pbpp;

	if (!Targa2Array("sample.tga", &data, &width, &height, &pbpp))
	{
		std::cout << "Can't read file";
		int a;
		std::cin >> a;
		return -1;
	}


	double d_matrix[N * N];
	double transposed[N * N];
	calculate_matrix(d_matrix);
	transpose_matrix(d_matrix, transposed);



	unsigned char quant[N * N] = {
		16, 11, 10, 16, 24, 40, 51, 61, 12, 12, 14, 19, 26, 58, 60, 55, 14, 13, 16, 24, 40, 57, 69, 56,
		14, 17, 22, 29, 51, 87, 80, 62, 18, 22, 37, 56, 68, 109, 103, 77, 24, 35, 55, 64, 81, 104, 113, 92,
		49, 64, 78, 87, 103, 121, 120, 101, 72, 92, 95, 98, 112, 100, 103, 99
	};


	int blocks_in_line = width / N;  //количество блоков в строке
	int blocks_in_column = height / N; //количество блоков в столбце

	int blocks = blocks_in_line * blocks_in_column; //общее количество блоков
	double* res = new double[width * height]; // матрица - результат косинусного преобразования

	auto time_start_chrono = std::chrono::high_resolution_clock::now();
	//ПРЯМОЕ
	//for (int i = 0; i < width * height; i++)
	//{
	//	data[i] = data[i] - 128;
	//}

	//ОДИН ЦИКЛ - ОДИН THREAD
	for (int kk = 0; kk < blocks; kk++)
	{
		double result[N * N];
		double result2[N * N];
		double buffer[N * N];

		for (int i = 0; i < N * N; i++)
		{
			result[i] = 0;
			result2[i] = 0;
			buffer[i] = 0;
		}


		int block_in_line = kk % blocks_in_line; //номер блока в строке, который будем обрабатывать (j)
		int block_in_column = kk / blocks_in_line; //номер в столбце (i)

		//ТУТ ПЕРЕПИСЫВАЕМ В БУФФЕР, ЧТОБЫ БЫЛО УДОБНЕЕ РАБОТАТЬ
		for (int row = 0; row < N; row++)
		{
			for (int col = 0; col < N; col++)
			{
				buffer[row * N + col] = data[(block_in_column * N + row) * width + block_in_line * N + col]-128;
			}
		}

		// D * F
		for (int row = 0; row < N; row++)
		{
			for (int col = 0; col < N; col++)
			{
				for (int inner = 0; inner < N; inner++)
				{
					result[row * N + col] += d_matrix[row * N + inner] * buffer[inner * N + col];
				}
			}
		}

		//(D * F) * Dt
		for (int row = 0; row < N; row++)
		{
			for (int col = 0; col < N; col++)
			{
				for (int inner = 0; inner < N; inner++)
				{
					result2[row * N + col] += result[row * N + inner] * transposed[inner * N + col];
				}
			}
		}

		//делим на матрицу квантования
		for (int i = 0; i < N; i++)
		{
			for (int j = 0; j < N; j++)
			{
				result2[i * N + j] = round(result2[i * N + j] / quant[i * N + j]);
			}
		}

		
		for (int row = 0; row < N; row++)
		{
			for (int col = 0; col < N; col++)
			{

				res[(block_in_column * N + row) * width + block_in_line * N + col] = result2[row * N + col];
				
			}
		}

	}


	//ОБРАТНОЕ
	unsigned char* res2 = new unsigned char[width * height];
	//ОДИН ЦИКЛ - ОДИН THREAD
	for (int kk = 0; kk < blocks; kk++)
	{
		double result[N * N];
		double result2[N * N];
		double buffer[N * N];
		for (int i = 0; i < N * N; i++)
		{
			result[i] = 0;
			result2[i] = 0;
			buffer[i] = 0;
		}

		int block_in_line = kk % blocks_in_line; //номер блока в строке, который будем обрабатывать (j)
		int block_in_column = kk / blocks_in_line; //номер в столбце (i)

		//ТУТ ПЕРЕПИСЫВАЕМ В БУФФЕР, ЧТОБЫ БЫЛО УДОБНЕЕ РАБОТАТЬ
		for (int row = 0; row < N; row++)
		{
			for (int col = 0; col < N; col++)
			{
				buffer[row * N + col] = res[(block_in_column * N + row) * width + block_in_line * N + col];
			}
		}

		//умножаем на матрицу квантования
		for (int i = 0; i < N; i++)
		{
			for (int j = 0; j < N; j++)
			{
				buffer[i * N + j] = buffer[i * N + j] * quant[i * N + j];
			}
		}

		//Dt*C
		for (int row = 0; row < N; row++)
		{
			for (int col = 0; col < N; col++)
			{
				for (int inner = 0; inner < N; inner++)
				{
					result[row * N + col] += transposed[row * N + inner] * buffer[inner * N + col];
				}
			}
		}

		//(Dt*C)*D
		for (int row = 0; row < N; row++)
		{
			for (int col = 0; col < N; col++)
			{
				for (int inner = 0; inner < N; inner++)
				{
					result2[row * N + col] += result[row * N + inner] * d_matrix[inner * N + col];
				}
			}
		}


		for (int row = 0; row < N; row++)
		{
			for (int col = 0; col < N; col++)
			{
				double tmp = result2[row * N + col]+128;
				if (tmp < 0)
					tmp = 0;
				if (tmp > 255)
					tmp = 255;
				res2[(block_in_column * N + row) * width + block_in_line * N + col] = (unsigned char)tmp;
			}
		}
	}
	//for (int i = 0; i < width * height; i++)
	//{
	//	res2[i] = res2[i] + 128;
	//}

	auto time_stop_chrono = std::chrono::high_resolution_clock::now();
	auto duration = std::chrono::duration_cast<std::chrono::microseconds>(time_stop_chrono - time_start_chrono).count();
	std::cout <<"Time is: "<< duration / 1000.0<<"\n";

	Array2Targa("result2.tga", res2, width, height, pbpp);
	return 0;
}

void transpose_matrix(double matrix[N * N], double tranposed[N * N])
{
	for (int i = 0; i < N; i++)
	{
		for (int j = 0; j < N; j++)
		{
			tranposed[j * N + i] = matrix[i * N + j];
		}
	}
}

void calculate_matrix(double matrix[N * N])
{
	const double PI = 3.1415926535897932384626433832795;
	double k0 = 1 / sqrt(N);
	double k = sqrt(2) / sqrt(N);
	for (int i = 0; i < N; i++)
	{
		for (int j = 0; j < N; j++)
		{
			double base = std::cos(j * (i + 0.5) * PI / N);
			if (j == 0)
			{
				matrix[i * N + j] = k0 * base;
			}
			else
			{
				matrix[i * N + j] = k * base;
			}
		}
	}
}
